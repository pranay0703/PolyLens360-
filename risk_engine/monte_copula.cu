#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hiprand/hiprand_kernel.h>

__global__ void simulate_kernel(double *d_results, int n_paths, int steps, double mu, double sigma, double s0) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_paths) return;
    hiprandState state;
    hiprand_init(42, idx, 0, &state);
    double dt = 1.0 / steps;
    double S = s0;
    for (int i = 0; i < steps; ++i) {
        double eps = hiprand_normal_double(&state) * sigma * sqrt(dt) + mu * dt;
        S *= exp(eps);
    }
    d_results[idx] = S;
}

extern "C" void run_mc(int n_paths, int steps, double mu, double sigma, double s0, double *h_out) {
    thrust::device_vector<double> d_results(n_paths);
    double *d_ptr = thrust::raw_pointer_cast(d_results.data());
    int block = 256;
    int grid  = (n_paths + block - 1) / block;
    simulate_kernel<<<grid, block>>>(d_ptr, n_paths, steps, mu, sigma, s0);
    thrust::host_vector<double> h_results = d_results;
    for (int i = 0; i < n_paths; ++i) {
        h_out[i] = h_results[i];
    }
}
